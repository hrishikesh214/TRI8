#include "hip/hip_runtime.h"


#include <iostream>
#include <numeric>
#include <stdio.h>
#include <stdlib.h>
using namespace std;

__global__ void max(int* input)
{
	const int tid = threadIdx.x; //similar to omp_get_thread_num()
	//we have spawned only one block see line no 46 <<< block, threads per block>>> //blockIdx.x gives block id starting from 0
	int iteration=1;
	auto int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0 )
	{
		printf("\n\titeration %d",iteration++);
		if (tid < number_of_threads) // still alive?
		{
			const auto int fst = tid * step_size * 2;
			const auto int snd = fst + step_size;
			if (input[fst] < input[snd])
				input[fst] = input[snd];
			printf("\nComparing and retain max element at arr[%d]= %d and arr[%d] = %d by thread id %d",fst,input[fst],snd,input[snd],tid);
		}

		step_size *= 2; 
		number_of_threads /= 2; 

	}
}

int main()
{
	const auto int count = 8;
	const int size = count * sizeof(int);
	int h[] = {1, 2, 3, 4, 5, 6, 7, 8};//10000

	int* d;
	
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	max <<<1, count / 2 >>>(d);

	int result;
	hipMemcpy(&result, d, sizeof(int), hipMemcpyDeviceToHost);

	cout << "MAX num is " << result << endl;


	hipFree(d);

	return 0;
}